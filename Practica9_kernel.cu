#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <random>
#include <stdio.h>
#include <iostream>
#include <time.h>

#define GPUErrorAssertion(ans) {gpuAssert((ans), __FILE__, __LINE__);};

inline void gpuAssert(hipError_t code, const char* file, int line, bool abort = true) {
    if (code != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n\r", hipGetErrorString(code), file, line);
        if (abort) exit(code);
    }
}

__global__ void search(int* a, int n, int* pos, int searchNum);


int main()
{
    const int n = 8;
    int searchNum = 5;
    int size = n * sizeof(int);
    
    int* a, * ans, *pos;

    // Allocate space for local variables
    a = (int*)malloc(size);
    ans = (int*)malloc(size);
    pos = (int*)malloc(sizeof(int));

    // Initialilze before the array
    pos[0] = -1;

    // Assign and print a random value to every position
    printf("Assigned values\n");
    for (int i = 0; i < n; i++) {
        a[i] = rand() % 256;
        printf("%d ", a[i]);
    }

    // Print separation line
    printf("\n");

    int* devA, *devPos;

    // Allocate Memory
    hipMalloc(&devA, size);
    hipMalloc(&devPos, sizeof(int));

    // Copy to GPU
    hipMemcpy(devA, a, size, hipMemcpyHostToDevice);
    hipMemcpy(devPos, pos, sizeof(int), hipMemcpyHostToDevice);

    // Declare grid
    dim3 block(1024);
    dim3 grid(n >= 1024 ? n / 1024 : 1);
    /*if (n >= 1024) {
        dim3 grid(n / 1024);
    }
    else {
        dim3 grid(1);
    }*/


    // Solve operations
    search << <grid, block >> > (devA, n, devPos, searchNum);
    hipDeviceSynchronize();
    hipMemcpy(pos, devPos, sizeof(int), hipMemcpyDeviceToHost);

    // Print solution
    if (pos[0] == -1) {
        printf("Element wasn't found\n");
    }
    else {
        printf("Element found at: %d postion\n", pos[0]);
    }


    // Clean
    hipDeviceSynchronize();
    hipDeviceReset();

    hipFree(devA);
    hipFree(devPos);

    return 0;
}

__global__ void search(int* a, int n, int* pos, int searchNum) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < n) {
        if (a[tid] == searchNum) {
            *pos = tid;
        }
    }
}
