#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <random>
#include <stdio.h>
#include <iostream>
#include <time.h>

using namespace std;


__global__ void convolution(int* a, int* k, int* c, int n, int m, int kernelSize) {
    int row = threadIdx.y;
    int col = threadIdx.x;

    int suma = 0;
    if (row > 0 && row < m - 1 && col>0 && col < n - 1) {
        for (int i = 0; i < kernelSize; i++) {
            for (int j = 0; j < kernelSize; j++) {
                //printf("%d\n", k[i * kernelSize + j]);
                suma += (a[(row-1) * m + i + (col - 1) + j] * k[i * kernelSize + j]);
                //printf("x: %d y: %d %d %d \n",col, row, a[i * m + j] , k[i * kernelSize + j]);
            }
        }
        c[row * m + col] = suma;
    }
}

int main() {

    const int n = 8, m = 8, kernelLength = 3;
    int* host_a, * host_c, *host_kernel;
    int* dev_a, * dev_c, * dev_kernel;
    host_a = (int*)malloc(n * m * sizeof(int));
    host_c = (int*)malloc(n * m * sizeof(int));
    host_kernel = (int*)malloc(kernelLength * kernelLength * sizeof(int));
    hipMalloc(&dev_a, n * m * sizeof(int));
    hipMalloc(&dev_c, n * m * sizeof(int));
    hipMalloc(&dev_kernel, kernelLength * kernelLength * sizeof(int));
    for (int i = 0; i < n * m; i++) {
        int r1 = (rand() % (3));
        host_a[i] = r1;
        host_c[i] = r1;
    }

    host_kernel[0] = 0;
    host_kernel[1] = 1;
    host_kernel[2] = 0;
    host_kernel[3] = 0;
    host_kernel[4] = 0;
    host_kernel[5] = 0;
    host_kernel[6] = 0;
    host_kernel[7] = 0;
    host_kernel[8] = 0;

    ///
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            printf("%d ", host_a[i * m + j]);
        }
        printf("\n");
    }
    ///

    hipMemcpy(dev_a, host_a, n * m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_c, host_c, n * m * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_kernel, host_kernel, kernelLength * kernelLength * sizeof(int), hipMemcpyHostToDevice);

    dim3 block(8, 8);
    convolution << <1, block >> > (dev_a, dev_kernel, dev_c, n, m, kernelLength);
    hipMemcpy(host_c, dev_c, n * m * sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();
    hipDeviceReset();

    cout << "Res:\n";
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < m; j++) {
            cout << host_c[i * n + j] << " ";
        }
        cout << "\n";
    }
    free(host_a);
    free(host_c);
    free(host_kernel);
    hipFree(dev_a);
    hipFree(dev_c);
    hipFree(dev_kernel);


    //return 0;
}
