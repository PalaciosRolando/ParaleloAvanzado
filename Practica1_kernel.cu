#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>

__global__ void pow(int *a, int *b, int *res){
  int i = threadIdx.x;
  res[i] = a[i] * b[i];
}

int main(){
  
  const int n = 3; 
  int size = n * sizeof(n);
  
  int a[n] = { 1, 7, 1 );
  int b[n] = { 4, 7, 1 );
  int res[n] = { 0, 0, 0 );
  
  int* devA = 0;
  int* devB = 0;
  int* devRes = 0;              
                
  hipMalloc((void**)&devA, size);
  hipMalloc((void**)&devB, size);
  hipMalloc((void**)&devRes, size);
    
  hipMemcpy(devA, a, size, cudaMempcyHostToDevice);
  hipMemcpy(devB, b, size, cudaMempcyHostToDevice);
  hipMemcpy(devRes, res, size, cudaMempcyHostToDevice);
  
  pow << <1, n >> > (devA, devB, devRes);
  hipDeviceSynchronize();

  hipMemcpy(res, devRes, size, hipMemcpyDeviceToHost); 

  printf("{%d, %d, %d}", res[0], res[1], res[2]);
  hipDeviceReset();
  
  hipFree(devA);
  hipFree(devB);
  hipFree(devRes);
  
  return 0;
}
